#include "hip/hip_runtime.h"
// Written by Vasily Volkov.
// Copyright (c) 2008-2009, The Regents of the University of California. 
// All rights reserved.

#include "codelets.h"

__global__ void IFFT512_device( float2 *work );

#define rank 8
__global__ void IFFT8_device_( float2 *work )
{	
    int tid = threadIdx.x;
    
    int bid = blockIdx.y * gridDim.x + blockIdx.x;
    int lo = bid & (4096/rank/64-1);
    int hi = bid &~(4096/rank/64-1);
    
    int i = lo*64 + tid;
    
    work += hi * (rank*64) + i;
    
    float2 a[rank];
    load<rank>( a, work, 512 );
    itwiddle_straight<rank>( a, i, 4096 );
    IFFT8( a );
    store<rank>( a, work, 512 );
}	

extern "C" void IFFT4096( float2 *work, int batch )
{	
    IFFT512_device<<< grid2D(batch*rank), 64 >>>( work );
    IFFT8_device_<<< grid2D(batch*(4096/rank)/64), 64 >>>( work );
}	
