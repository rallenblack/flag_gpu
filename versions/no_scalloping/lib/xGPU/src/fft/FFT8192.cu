#include "hip/hip_runtime.h"
// Written by Vasily Volkov.
// Copyright (c) 2008-2009, The Regents of the University of California. 
// All rights reserved.

#include "codelets.h"

__global__ void FFT512_device( float2 *work );

#define rank 16
__global__ void FFT16_device_( float2 *work )
{	
    int tid = threadIdx.x;

    int bid = blockIdx.y * gridDim.x + blockIdx.x;
    int lo = bid & (8192/rank/64-1);
    int hi = bid &~(8192/rank/64-1);

    int i = lo*64 + tid;
    
    work += hi * (rank*64) + i;
    
    float2 a[rank];
    load<rank>( a, work, 512 );
    FFT16( a );
    twiddle<rank>( a, i, 8192 );
    store<rank>( a, work, 512 );
}	

extern "C" void FFT8192( float2 *work, int batch )
{	
    FFT16_device_<<< grid2D(batch*(8192/rank)/64), 64 >>>( work );
    FFT512_device<<< grid2D(batch*rank), 64 >>>( work );
}	
