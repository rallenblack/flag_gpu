#include "hip/hip_runtime.h"
// Written by Vasily Volkov.
// Copyright (c) 2008-2009, The Regents of the University of California. 
// All rights reserved.

#include "codelets.h"

__global__ void FFT16_device( float2 *dst, float2 *src )
{	
    int tid = threadIdx.x;
    
    int iblock = blockIdx.y * gridDim.x + blockIdx.x;
    int index = iblock * 1024 + tid;
    src += index;
    dst += index;
    
    float2 a[16];
    
    load<16>( a, src, 64 );

    FFT16( a );

    store<16>( a, dst, 64 );
}	
    
extern "C" void FFT16( float2 *work, int batch )
{	
    FFT16_device<<< grid2D(batch/64), 64 >>>( work, work );
}	
