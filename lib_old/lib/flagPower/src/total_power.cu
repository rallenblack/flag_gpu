#include "hip/hip_runtime.h"
#include "total_power.h"

__global__ void total_power_kernel1(unsigned char * input, float * output, int start_t) {

    // Declare dynamic shared memory
    __shared__ float power[nblocks1];

    // Get indicies
    int c = blockIdx.x;
    int f = blockIdx.y;
    int t = threadIdx.x;
    
    // Get internal index (internal to block)
    int sid = t;

    // Get absolute index
    int idx = f + 8*NF*c + 8*NF*NC*t;

    if (sid + start_t < NM*NT) {
    	// Extract real and imaginary components;
    	float real = (float)input[2*idx];
    	float imag = (float)input[2*idx + 1];

    	// Compute instantaneous power
    	power[sid] = real*real + imag*imag;
    }
    else {
    	power[sid] = 0.0;
    }

    // Complete power computation before moving on
    __syncthreads();

    // Perform reduction
    for (int s = blockDim.x/2; s > 0; s>>=1) {
        if (sid < s) {
            power[sid] += power[sid + s];
        }
        __syncthreads();
    }

    // Save sum to output
    if (sid == 0) {
        output[f + 8*NF*c] = power[0];
    }
}



__global__ void total_power_kernel2(float * input, float * output) {

    // Declare dynamic shared memory
    __shared__ float power[nblocks2];

    // Get indicies
    int c = blockIdx.x;
    int f = blockIdx.y;
    int t = threadIdx.x;
    
    // Get internal index (internal to block)
    int sid = t;

    // Get absolute index
    int idx = f + 8*NF*c + 8*NF*NC*t;

    power[sid] = input[idx];

    // Complete power computation before moving on
    __syncthreads();

    // Perform reduction
    for (int s = blockDim.x/2; s > 0; s>>=1) {
        if (sid < s) {
            power[sid] += power[sid + s];
        }
        __syncthreads();
    }

    // Save sum to output
    if (sid == 0) {
        output[f + 8*NF*c] = power[0];
    }
}



__global__ void total_power_kernel3(float * input, float * output) {

    // Declare dynamic shared memory
    __shared__ float power[pow2];
    
    // Get indices
    int f = blockIdx.x;
    int c = threadIdx.x;

    // Get internal index (internal to block)
    int sid = c;

    // Get absolute index
    int idx = f + 8*NF*c;

    if (sid < NC) {
    	// Copy input to shared memory
    	power[sid] = input[idx];
    }
    else {
    	power[sid] = 0.0;
    }

    // Finish copy before proceeding
    __syncthreads();

    // Perform reduction
    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        if (sid < s) {
            power[sid] += power[sid + s];
        }
        __syncthreads();
    }

    // Save sum to input
    if (sid == 0) {
        output[f] = power[0];
    }
}

static unsigned char * d_input;
static float * d_output1;
static float * d_output2;
static float * d_output3;

void initTotalPower() {
    hipMalloc((void **) &d_input, 8*NF*NC*NT*NM*sizeof(unsigned char)*2);
    hipMalloc((void **) &d_output1, 8*NF*NC*nblocks2*sizeof(float));
    hipMalloc((void **) &d_output2, 8*NF*NC*sizeof(float));
    hipMalloc((void **) &d_output3, 8*NF*sizeof(float));
}

void getTotalPower(unsigned char * input, float * output) {

    hipMemcpy(d_input, input, 8*NF*NC*NT*NM*2*sizeof(unsigned char), hipMemcpyHostToDevice);

    /**********************************************
     * Reduce over 1024 time samples
     **********************************************/
    dim3 gridSize1(NC,8*NF,1);
    dim3 blockSize1(nblocks1,1,1);
    hipStream_t s[nblocks2];
    for (int i = 0; i < nblocks2; i++) {
        hipStreamCreate(&s[i]);
    }
    
    for (int i = 0; i < nblocks2; i++) {
        // int in_off = 8*NF*NC*nblocks1*i;
        // int out_off = 8*NF*NC*i;
        total_power_kernel1<<<gridSize1, blockSize1, 0, s[i]>>>(d_input+NA*NC*nblocks1*i, d_output1+NA*NC*i, nblocks1*i);
    }
    hipDeviceSynchronize();
    hipError_t ret = hipGetLastError();
    if (ret != hipSuccess) {
        printf("ERROR: total_power_kernel1 - %s\n", hipGetErrorString(ret));
    }




    /**********************************************
     * Reduce over remaining time samples
     **********************************************/
    dim3 gridSize2(NC,8*NF,1);
    dim3 blockSize2(nblocks2,1,1);
    total_power_kernel2<<<gridSize2, blockSize2>>>(d_output1, d_output2);
    ret = hipGetLastError();
    if (ret != hipSuccess) {
        printf("ERROR: total_power_kernel2 - %s\n", hipGetErrorString(ret));
    }


    /**********************************************
     * Reduce over frequency channels
     **********************************************/
    dim3 gridSize3(8*NF,1,1);
    dim3 blockSize3(pow2,1,1);

    total_power_kernel3<<<gridSize3, blockSize3>>>(d_output2, d_output3);
    ret = hipGetLastError();
    if (ret != hipSuccess) {
        printf("ERROR: total_power_kernel3 - %s\n", hipGetErrorString(ret));
    }

    hipMemcpy(output, d_output3, 8*NF*sizeof(float), hipMemcpyDeviceToHost);
}
