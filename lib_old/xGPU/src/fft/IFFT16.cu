#include "hip/hip_runtime.h"
// Written by Vasily Volkov.
// Copyright (c) 2008-2009, The Regents of the University of California. 
// All rights reserved.

#include "codelets.h"

__global__ void IFFT16_device( float2 *dst, float2 *src )
{	
    int tid = threadIdx.x;
    
    int iblock = blockIdx.y * gridDim.x + blockIdx.x;
    int index = iblock * 1024 + tid;
    src += index;
    dst += index;
    
    float2 a[16];
    
    load<16>( a, src, 64 );

    IFFT16( a );

    store<16>( a, dst, 64 );
}	
    
extern "C" void IFFT16( float2 *work, int batch )
{	
    IFFT16_device<<< grid2D(batch/64), 64 >>>( work, work );
}	
